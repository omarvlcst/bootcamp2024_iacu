
#include <hip/hip_runtime.h>
#include <stdio.h>

// Ciclo que enumera threads desde el 0 hasta el n-1
__global__ void unkernel(){
    printf("Hola desde el bloque %d thread: %d\n", blockIdx.x, threadIdx.x); //threadIdx.x es un identificador de cada thread
}

int main(void){
    unkernel<<<1,10>>>();
    hipDeviceSynchronize();
    return 0;
}

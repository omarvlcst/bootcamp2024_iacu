
#include <hip/hip_runtime.h>
#include <stdio.h>

// PROGRAMA PARA MULTIPLICAR DOS NUMEROS a y b Y REGRESARLOS COMO UN NUMERO c

// El kernel de CUDA
__global__ void muln(float* a, float* b, float* c){
  *c = (*a)*(*b);
}

int main(void){
  float a,b,c;
  float* d_a; float* d_b; float* d_c;

  int tamf = sizeof(float);

  // Reservando memoria en el device para a,b,c,
  hipMalloc((void **) &d_a, tamf);
  hipMalloc((void **) &d_b, tamf);
  hipMalloc((void **) &d_c, tamf);

  a = 2.0;
  b = 5.2;

  // Copiando al device
  hipMemcpy(d_a, &a, tamf, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, tamf, hipMemcpyHostToDevice);

  // Lanzando el kernel
  muln<<<1,1>>>(d_a, d_b, d_c);

  // Recuperando el device
  hipMemcpy(&c, d_c, tamf, hipMemcpyDeviceToHost);

  // Limpiando memoria en el device
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  printf("%f\n",c);
  return 0;
}

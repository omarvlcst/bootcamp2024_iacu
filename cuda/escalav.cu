
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void escalav(float* esc, float *vect){
    vect[threadIdx.x] = *esc * vect[threadIdx.x];
} 

int main(void){
  // Datos en el host (la GPU)
  float a[10] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
  float esc=4.5;

  int tamv=10;
  int tamd=sizeof(float);

  // Las variables del device (la tarjeta grafica)
  float* d_a;
  float* d_esc;
  //float* d_tamd;

  hipMalloc((void **) &d_a, tamv*tamd);
  hipMalloc((void **) &d_esc, tamd);
  //cudaMalloc((void **) tamv);
  hipMemcpy(d_a, a, tamv*tamd, hipMemcpyHostToDevice);
  hipMemcpy(d_esc, &esc, tamd, hipMemcpyHostToDevice);

  escalav<<<1,10>>>(d_esc, d_a);

  hipMemcpy(a, d_a, tamv*tamd, hipMemcpyDeviceToHost);
  hipFree(d_a), hipFree(d_esc);

  printf("%f\n", a[0]); //Para imprimir el vector completo con cada entrada, hay que escribir un ciclo FOR que imprima todos
  return 0;
}
